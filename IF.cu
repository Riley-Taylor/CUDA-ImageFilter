//Riley Taylor-S00348849
//CSCI4250, HW2
//Image Filtering


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
//Thanks for showing us this!
#define iceil(num,den) (num+den-1)/den

//Kernel Function
__global__ void imgFilKernel(float* d_img_in, float* d_img_out, int *d_img_filter,int w, int h){
	
	//Access each "pixel" array index!
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	//The if statement helps in creating a small border so the image does not go out of bounds!
	if(col < w && row < h && row > 0 && col > 0 && row < h-1 && col < w-1)																	//4 5 3 7 1 8 0 6 2
		d_img_out[row*w + col] = ((((((((((d_img_in[row*w + col] * d_img_filter[4]) + d_img_in[row*w + (col+1)] * d_img_filter[5]) + d_img_in[row*w + (col-1)] * d_img_filter[3]) +d_img_in[(row+1)*w + col] * d_img_filter[7]) + d_img_in[(row-1)*w + col] * d_img_filter[1]) + d_img_in[(row+1)*w + (col+1)] * d_img_filter[8]) + d_img_in[(row-1)*w + (col-1)] * d_img_filter[0]) +d_img_in[(row+1)*w + (col-1)] * d_img_filter[6]) +d_img_in[(row-1)*w + (col+1)] * d_img_filter[2])/9);
	//while this is not the best way to filter (index by index) it works for all examples you just have to edit the numbers in the filter
}

void imgFil(float*img_in, float* img_out, int *img_filter, int w, int h) {
	
	//This number of bytes are going be allocated and transferred
	int size = w * h * sizeof(float);
	int size2 = 8*sizeof(int);
	
	float *d_img_in, *d_img_out;
	int *d_img_filter;
	//GPU memory allocation!
	hipMalloc((void**)&d_img_in, size);
	hipMalloc((void**)&d_img_out, size);
	hipMalloc((void**)&d_img_filter,size2);
	
	//GPU data transfer
	hipMemcpy(d_img_in, img_in, size, hipMemcpyHostToDevice);
	hipMemcpy(d_img_filter,img_filter,size2,hipMemcpyHostToDevice);
	dim3 myBlockDim(16, 16, 1);
	dim3 myGridDim(iceil(w, 16), iceil(h, 16), 1);
	//Call the kernel!
	imgFilKernel <<<myGridDim, myBlockDim >>> (d_img_in, d_img_out, d_img_filter, w, h);

	//Transfer "Image" back to HOST!
	hipMemcpy(img_out, d_img_out, size, hipMemcpyDeviceToHost);

	//free the allocated memory
	hipFree(d_img_in);
	hipFree(d_img_out);
	hipFree(d_img_filter);
}

//Prints the image!
void printImage(float* img, int w, int h) {
	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			cout << img[i*w + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

int main(){

	int w = 30; int h = 20;
	//allocating CPU memory!
	float *img_in = new float[w*h];
	float *img_out = new float[w*h];
	int *img_filter = new int[8];
	//Fill array with 3s
	for (int i = 0; i < h; i++)
		for (int j = 0; j < w; j++)
			img_in[i*w + j] = 3;
	//fill the filter with 9s
	for (int i = 0; i < 9; i++)
		img_filter[i] = 9;
	
	//prints the "image" before hand so you can see the difference.
	printImage(img_in, w, h);

	//apply filter to "image"
	imgFil(img_in, img_out, img_filter, w, h);
	
	//THis prints out the filter!
	for (int x=0; x<1; x++)
		for (int y=0; y<3; y++)
		{
			cout << img_filter[x*x+y]<<img_filter[x*x+y]<<img_filter[x*x+y];
			cout <<endl;
		}		
	
	cout <<endl;
	//prints the after "image" so you can see the applied filter
	printImage(img_out, w, h);

	return 0;
}
